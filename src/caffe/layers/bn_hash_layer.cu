#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/bn_hash_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe\util\HashData.h"

namespace caffe {

	template <typename Dtype>
	__global__ void hash2temp_kernel(const Dtype *hash, const int* validPos, int m_bar,
		int channels, int def_num, int total_def_num, Dtype *temp)
	{
		const int m = m_bar * m_bar * m_bar;
		CUDA_KERNEL_LOOP(valid_v_channels, def_num * channels)
		{
			const int c = valid_v_channels / def_num;
			const int valid_v = valid_v_channels - c * def_num;
			const int v = validPos[valid_v];
			temp[valid_v + c * total_def_num] = hash[v + c * m];
		}
	}

	template <typename Dtype>
	__global__ void temp2hash_kernel(Dtype *hash, const int* validPos, int m_bar,
		int channels, int def_num, int total_def_num, const Dtype *temp)
	{
		const int m = m_bar * m_bar * m_bar;
		CUDA_KERNEL_LOOP(valid_v_channels, def_num*channels)
		{
			const int c = valid_v_channels / def_num;
			const int valid_v = valid_v_channels - c * def_num;
			const int v = validPos[valid_v];
			hash[v + c * m] = temp[valid_v + c * total_def_num];
		}
	}

	template <typename Dtype>
	__global__ void substract_mean_kernel(Dtype* temp, int channels, int total_defNum, const Dtype* mean)
	{
		CUDA_KERNEL_LOOP(index, channels*total_defNum)
		{
			const int c = index / total_defNum;
			temp[index] -= mean[c];
		}
	}

	template <typename Dtype>
	__global__ void inv_sqrt_eps_var_kernel(Dtype* temp, int channels, int total_defNum, const Dtype* var, Dtype eps)
	{
		CUDA_KERNEL_LOOP(index, channels*total_defNum)
		{
			const int c = index / total_defNum;
			temp[index] /= sqrt(var[c] + eps);
		}
	}

	template <typename Dtype>
	void BNHashLayer<Dtype>::forward_hash2temp_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top)
	{
		const Dtype *hash = bottom[HASH_DATA_BLOB]->gpu_data();
		const int *validPos = (const int*)bottom[VALID_POS_BLOB]->gpu_data();
		Dtype* temp = temp_.mutable_gpu_data();
		const int batch_num = bottom[M_BAR_BLOB]->shape(0);
		const int total_def_num = temp_.shape(1);
		for (int i = 0; i < batch_num; ++i)
		{
			const int m_bar = (int)bottom[M_BAR_BLOB]->cpu_data()[i];
			const int def_num = bottom[DEFNUM_BLOB]->cpu_data()[i];

			hash2temp_kernel << <CAFFE_GET_BLOCKS(def_num*channels_), CAFFE_CUDA_NUM_THREADS >> > (
				hash, validPos, m_bar, channels_, def_num, total_def_num, temp
				);

			//to next hash
			const int m = m_bar * m_bar * m_bar;
			hash += m * channels_;
			validPos += m;
			temp += def_num;
		}
	}

	template <typename Dtype>
	void BNHashLayer<Dtype>::forward_temp2hash_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top)
	{
		Dtype *hash = (Dtype*)top[HASH_DATA_BLOB]->mutable_gpu_data();
		const int *validPos = (const int*)bottom[VALID_POS_BLOB]->gpu_data();
		const Dtype* temp = temp_.gpu_data();
		const int batch_num = bottom[M_BAR_BLOB]->shape(0);
		const int total_def_num = temp_.shape(1);
		for (int i = 0; i < batch_num; ++i)
		{
			const int m_bar = (int)bottom[M_BAR_BLOB]->cpu_data()[i];
			const int def_num = bottom[DEFNUM_BLOB]->cpu_data()[i];

			temp2hash_kernel << <CAFFE_GET_BLOCKS(def_num*channels_), CAFFE_CUDA_NUM_THREADS >> > (
				hash, validPos, m_bar, channels_, def_num, total_def_num, temp
				);

			//to next hash
			const int m = m_bar * m_bar * m_bar;
			hash += m * channels_;
			validPos += m;
			temp += def_num;
		}
	}

	template <typename Dtype>
	void BNHashLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top)
	{
		//total num
		const int total_defNum = temp_.shape(1);
		const Dtype mean_div = Dtype(1) / Dtype(total_defNum);
		//const Dtype var_div = Dtype(1) / Dtype(std::max(1, total_defNum - 1));
		const Dtype var_div = mean_div;	//will be bias-corrected when adding to blob[1]

		// prepare temp_ array
		forward_hash2temp_gpu(bottom, top);

		if (use_global_stats_) 
		{
			// use the stored mean/variance estimates.
			const Dtype scale_factor = this->blobs_[2]->cpu_data()[0] == 0 ?
				0 : 1 / this->blobs_[2]->cpu_data()[0];
			caffe_gpu_scale(variance_.count(), scale_factor,
				this->blobs_[0]->gpu_data(), mean_.mutable_gpu_data());
			caffe_gpu_scale(variance_.count(), scale_factor,
				this->blobs_[1]->gpu_data(), variance_.mutable_gpu_data());
		}
		else
		{
			/********1. compute the mean EX for each channel *************/
			caffe_gpu_gemv(CblasNoTrans, channels_, total_defNum, mean_div, 
				temp_.gpu_data(), mean_multiplier_.gpu_data(), Dtype(0), mean_.mutable_gpu_data());
		}

		/**********************2 substract mean****************/
		substract_mean_kernel << <CAFFE_GET_BLOCKS(channels_*total_defNum), CAFFE_CUDA_NUM_THREADS >> > (
			temp_.mutable_gpu_data(), channels_, total_defNum, mean_.gpu_data()
			);

		/********************3. compute variance using var(X) = E((X-EX)^2)***********************/
		if (!use_global_stats_)
		{
			caffe_gpu_mul(temp_.count(), temp_.gpu_data(), temp_.gpu_data(), temp2_.mutable_gpu_data());
			caffe_gpu_gemv(CblasNoTrans, channels_, total_defNum, var_div,
				temp2_.gpu_data(), mean_multiplier_.gpu_data(), Dtype(0), variance_.mutable_gpu_data());

			// compute and save moving average
			this->blobs_[2]->mutable_cpu_data()[0] *= moving_average_fraction_;
			this->blobs_[2]->mutable_cpu_data()[0] += 1;

			caffe_gpu_axpby(mean_.count(), Dtype(1), mean_.gpu_data(),
				moving_average_fraction_, this->blobs_[0]->mutable_gpu_data());

			Dtype bias_correction_factor = total_defNum > 1 ? Dtype(total_defNum) / (total_defNum - 1) : 1;
			caffe_gpu_axpby(variance_.count(), bias_correction_factor,
				variance_.gpu_data(), moving_average_fraction_, this->blobs_[1]->mutable_gpu_data());

		}

		/********************4. compute final top (X-mean(X))/(sqrt(var(X)+eps))***********************/
		// normalize variance
		// div by sqrt(var(X)+eps)
		inv_sqrt_eps_var_kernel << <CAFFE_GET_BLOCKS(channels_*total_defNum), CAFFE_CUDA_NUM_THREADS >> > (
			temp_.mutable_gpu_data(), channels_, total_defNum, variance_.gpu_data(), eps_
			);

		forward_temp2hash_gpu(bottom, top);
		
		caffe_copy(bottom[CHANNEL_BLOB]->count(), bottom[CHANNEL_BLOB]->cpu_data(),
			top[CHANNEL_BLOB]->mutable_cpu_data());
		caffe_copy(bottom[DENSE_RES_BLOB]->count(), bottom[DENSE_RES_BLOB]->cpu_data(),
			top[DENSE_RES_BLOB]->mutable_cpu_data());
	}

	template <typename Dtype>
	void BNHashLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
	{
		const Dtype* top_diff = top[HASH_DATA_BLOB]->gpu_diff();
		Dtype* bottom_diff = bottom[HASH_DATA_BLOB]->mutable_gpu_diff();
		const int total_defNum = temp_.shape(1);
		const Dtype mean_div = Dtype(1) / Dtype(total_defNum);

		//convert top_dif to tmp
		backward_topDif2temp_gpu(bottom, top);
		if (use_global_stats_)
		{
			// replicate inv_variance to input size
			caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, channels_, total_defNum, 1,
				(Dtype)1, inv_sqrt_var_.gpu_data(), mean_multiplier_.gpu_data(), (Dtype)0,
				temp2_.mutable_gpu_data());
			caffe_gpu_mul(temp_.count(), temp_.gpu_data(), temp2_.gpu_data(), temp_.mutable_gpu_data());
			backward_temp2BottomDif_gpu(bottom, top);
			return;
		}


		// if Y = (X-mean(X))/(sqrt(var(X)+eps)), then
		//
		// dE(Y)/dX =
		//   (dE/dY - mean(dE/dY) - mean(dE/dY \cdot Y) \cdot Y)
		//     ./ sqrt(var(X) + eps)
		//
		// where \cdot and ./ are hadamard product and elementwise division,
		// respectively, dE/dY is the top diff, and mean/var/sum are all computed
		// along all dimensions except the channels dimension.  In the above
		// equation, the operations allow for expansion (i.e. broadcast) along all
		// dimensions except the channels dimension where required.
		// --------------------------------------------------------
		// If disable_vairance is set, the derivative change to
		// dE(Y)/dX = dE/dY - mean(dE/dY)
		// If disable_mean is set, derivative becomes
		// dE(Y)/dX =
		//   (dE/dY - mean(dE/dY \cdot Y) \cdot Y)
		//     ./ sqrt(var(X) + eps)

		//step1. mean(dE/dY \cdot Y)
		top_2_buf_gpu(bottom, top, temp2_);	//convert Y to temp2_
		//dE/dY \cdot Y; // NOTE: here temp_ is modified
		caffe_gpu_mul(temp_.count(), temp_.gpu_data(), temp2_.gpu_data(), temp_.mutable_gpu_data());
		//mean
		caffe_gpu_gemv(CblasNoTrans, channels_, total_defNum, mean_div,
			temp_.gpu_data(), mean_multiplier_.gpu_data(), Dtype(0), mean_.mutable_gpu_data());

		//step2. mean(dE/dY \cdot Y) \cdot Y
		//reshape mean to input size
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, channels_, total_defNum, 1,
			(Dtype)1, mean_.gpu_data(), mean_multiplier_.gpu_data(), (Dtype)0,
			temp_.mutable_gpu_data());
		// mean(dE/dY \cdot Y) \cdot Y
		caffe_gpu_mul(temp_.count(), temp_.gpu_data(), temp2_.gpu_data(), temp2_.mutable_gpu_data());

		//step3. dE/dY - mean(dE/dY) - mean(dE/dY \cdot Y) \cdot Y
		//convert top_dif to tmp
		backward_topDif2temp_gpu(bottom, top);
		//mean(dE/dY)
		caffe_gpu_gemv(CblasNoTrans, channels_, total_defNum, mean_div,
			temp_.gpu_data(), mean_multiplier_.gpu_data(), Dtype(0), mean_.mutable_gpu_data());

		//dE/dY - mean(dE/dY \cdot Y) \cdot Y
		caffe_gpu_sub(temp_.count(), temp_.gpu_data(), temp2_.gpu_data(), temp_.mutable_gpu_data());

		//-= mean(dE/dY)
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, channels_,
			total_defNum, 1, (Dtype)-1, mean_.gpu_data(), mean_multiplier_.gpu_data(),
			(Dtype)1, temp_.mutable_gpu_data());

		//step4. (dE/dY - mean(dE/dY) - mean(dE/dY \cdot Y) \cdot Y) / sqrt(var(X) + eps)
		// replicate inv_variance to input size
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, channels_, total_defNum, 1,
			(Dtype)1, inv_sqrt_var_.gpu_data(), mean_multiplier_.gpu_data(), (Dtype)0,
			temp2_.mutable_gpu_data());
		caffe_gpu_mul(temp_.count(), temp_.gpu_data(), temp2_.gpu_data(), temp_.mutable_gpu_data());
		backward_temp2BottomDif_gpu(bottom, top);
	}

	template <typename Dtype>
	void BNHashLayer<Dtype>::backward_topDif2temp_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top)
	{
		const Dtype *hash_dif = top[HASH_DATA_BLOB]->gpu_diff();
		const int *validPos = (const int*)bottom[VALID_POS_BLOB]->gpu_data();
		Dtype* temp = temp_.mutable_gpu_data();
		const int batch_num = (int)bottom[M_BAR_BLOB]->shape(0);
		const int total_def_num = temp_.shape(1);
		for (int i = 0; i < batch_num; ++i)
		{
			const int m_bar = (int)bottom[M_BAR_BLOB]->cpu_data()[i];
			const int def_num = bottom[DEFNUM_BLOB]->cpu_data()[i];
			const int m = m_bar * m_bar * m_bar;

			hash2temp_kernel << <CAFFE_GET_BLOCKS(def_num*channels_), CAFFE_CUDA_NUM_THREADS >> > (
				hash_dif, validPos, m_bar, channels_, def_num, total_def_num, temp
				);

			//to next hash
			hash_dif += m * channels_;
			validPos += m;
			temp += def_num;
		}
	}

	template <typename Dtype>
	void BNHashLayer<Dtype>::backward_temp2BottomDif_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top)
	{
		Dtype *hash_dif = bottom[HASH_DATA_BLOB]->mutable_gpu_diff();
		const int *validPos = (const int*)bottom[VALID_POS_BLOB]->gpu_data();
		const Dtype* temp = temp_.gpu_data();
		const int batch_num = (int)bottom[M_BAR_BLOB]->shape(0);
		const int total_def_num = temp_.shape(1);
		for (int i = 0; i < batch_num; ++i)
		{
			const int m_bar = (int)bottom[M_BAR_BLOB]->cpu_data()[i];
			const int def_num = bottom[DEFNUM_BLOB]->cpu_data()[i];
			const int m = m_bar * m_bar * m_bar;

			temp2hash_kernel << <CAFFE_GET_BLOCKS(def_num*channels_), CAFFE_CUDA_NUM_THREADS >> > (
				hash_dif, validPos, m_bar, channels_, def_num, total_def_num, temp
				);

			//to next hash
			hash_dif += m * channels_;
			validPos += m;
			temp += def_num;
		}
	}

	template <typename Dtype>
	void BNHashLayer<Dtype>::top_2_buf_gpu(const vector<Blob<Dtype>*>& bottom, 
		const vector<Blob<Dtype>*>& top, Blob<Dtype> &buf)
	{
		const Dtype *hash = top[HASH_DATA_BLOB]->gpu_data();
		const int *validPos = (const int*)bottom[VALID_POS_BLOB]->gpu_data();
		Dtype* buf_ptr = buf.mutable_gpu_data();
		const int batch_num = (int)bottom[M_BAR_BLOB]->shape(0);
		const int total_def_num = buf.shape(1);
		for (int i = 0; i < batch_num; ++i)
		{
			const int m_bar = (int)bottom[M_BAR_BLOB]->cpu_data()[i];
			const int def_num = bottom[DEFNUM_BLOB]->cpu_data()[i];
			const int m = m_bar * m_bar * m_bar;

			hash2temp_kernel << <CAFFE_GET_BLOCKS(def_num*channels_), CAFFE_CUDA_NUM_THREADS >> > (
				hash, validPos, m_bar, channels_, def_num, total_def_num, buf_ptr
				);

			//to next hash
			hash += m * channels_;
			validPos += m;
			buf_ptr += def_num;
		}
	}

	INSTANTIATE_LAYER_GPU_FUNCS(BNHashLayer);
}  // namespace caffe
